#include "hip/hip_runtime.h"
// #include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <opencv2/opencv.hpp>
#include <bits/stdc++.h>
#include <sys/time.h>

using namespace cv;
using namespace std;

#define WIDTH 720
#define HEIGHT 480
#define CHANNELS 3
#define ITERATIONS 20

//function to get time
#ifndef TIMER_H
#define TIMER_H

typedef unsigned long long timestamp_t;

static timestamp_t

get_timestamp ()
{
  struct timeval now;
  gettimeofday (&now, NULL);
  return  now.tv_usec + (timestamp_t)now.tv_sec * 1000000;
}
#endif



__global__ void nearest_neighbour_scaling(
    unsigned char *input_image, 
    unsigned char *output_image,
    int width_input, 
    int height_input,
    int width_output, 
    int height_output,
    int channels) {
    const float x_ratio = (width_input + 0.0) / width_output;
    const float y_ratio = (height_input + 0.0) / height_output;

	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    int px = 0, py = 0; 
    const int input_width_step = width_input * channels;
    const int output_width_step = width_output * channels;

    if ((xIndex < width_output) && (yIndex < height_output)){
        py = ceil(yIndex * y_ratio);
        px = ceil(xIndex * x_ratio);
        for (int channel = 0; channel < channels; channel++){
            *(output_image + (yIndex * output_width_step + xIndex * channels + channel)) =  *(input_image + (py * input_width_step + px * channels + channel));
        }
    }
}



int main(int argc, char* argv[]) {
    
    const string source_image_path = argv[1];
    const string result_image_path = argv[2];
    const int threads = atoi(argv[3]);


    Mat output_image(HEIGHT, WIDTH, CV_8UC3); 
    Mat input_image = imread(source_image_path);
    timestamp_t start_a, end_a;
    double avg;
    
    hipEvent_t start, end;
    
    const int input_bytes = input_image.cols * input_image.rows * input_image.channels() * sizeof(unsigned char);
    const int output_bytes = output_image.cols * output_image.rows * output_image.channels() * sizeof(unsigned char);

    unsigned char *d_input, *d_output;
    hipMalloc<unsigned char>(&d_input, input_bytes);
    hipMalloc<unsigned char>(&d_output, output_bytes);

    hipMemcpy(d_input, input_image.ptr(), input_bytes, hipMemcpyHostToDevice);
    hipEventCreate(&start);
    hipEventCreate(&end);

    
    int width_input = input_image.cols;
    int height_input = input_image.rows;
    int channels_input = input_image.channels();
    int width_output = output_image.cols;
    int height_output = output_image.rows;
    int channels_output = output_image.channels();

    hipEventRecord(start, NULL);
    start_a = get_timestamp();
    const dim3 threadsPerBlock(threads, threads);
    const dim3 numBlocks(width_output / threadsPerBlock.x, height_output / threadsPerBlock.y);
    for(int i = 0; i < ITERATIONS; i++){
            nearest_neighbour_scaling<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width_input, height_input, width_output, height_output, CHANNELS);
    }
    end_a = get_timestamp();
    hipEventRecord(end, NULL);
    hipEventSynchronize(end);
    avg = (end_a - start_a);
    printf("%f\n",avg/(double)1000);
    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, end);
    float secPerMatrixMul = msecTotal / (ITERATIONS * 1.0f);
    printf(
        "Time= %.8f ms",
        secPerMatrixMul
    );

  
    hipMemcpy(output_image.ptr(), d_output, output_bytes, hipMemcpyDeviceToHost);

    imwrite(result_image_path, output_image);

    hipFree(d_input);
    hipFree(d_output);

    printf("Done\n");
    return 0;
}
